#include "hip/hip_runtime.h"
#include "./common.cpp"

int matsize = 100;

__global__ void kernelMatMul( float* C, float* A, float* B, int matsize ) {
	int gy = blockIdx.y * blockDim.y + threadIdx.y; 
	int gx = blockIdx.x * blockDim.x + threadIdx.x; 
	if (gy < matsize && gx < matsize)
	{
		float sum = 0.0f;
		for (int k = 0; k < matsize; ++k) {
			int idxA = gy * matsize + k;
			int idxB = k * matsize + gx;
			sum += A[idxA] * B[idxB];
		}
		int idxC = gy * matsize + gx;
		C[idxC] = sum;
	}
}
int main(void) {
	float* matA = nullptr;
	float* matB = nullptr;
	float* matC = nullptr;
	matA = new float[matsize * matsize];
	matB = new float[matsize * matsize];
	matC = new float[matsize * matsize];
	srand( 0 );
	setNormalizedRandomData( matA, matsize * matsize );
	setNormalizedRandomData( matB, matsize * matsize );
	float* dev_matA = nullptr;
	float* dev_matB = nullptr;
	float* dev_matC = nullptr;
	hipMalloc( (void**)&dev_matA, matsize *  matsize * sizeof(float) );
	hipMalloc( (void**)&dev_matB,  matsize *  matsize * sizeof(float) );
	hipMalloc( (void**)&dev_matC,  matsize *  matsize * sizeof(float));
	hipMemcpy( dev_matA, matA,  matsize * matsize * sizeof(float),  hipMemcpyHostToDevice);
	hipMemcpy( dev_matB, matB,  matsize * matsize * sizeof(float), hipMemcpyHostToDevice);
	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid((matsize + dimBlock.x - 1) / dimBlock.x, (matsize + dimBlock.y - 1) / dimBlock.y, 1);
	ELAPSED_TIME_BEGIN(0);
	kernelMatMul <<< dimGrid, dimBlock>>>( dev_matC, dev_matA, dev_matB, matsize);
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	hipMemcpy( matC, dev_matC, matsize *matsize * sizeof(float), hipMemcpyDeviceToHost);
	hipFree( dev_matA );
	hipFree( dev_matB );
	hipFree( dev_matC );
	printf("matrix size = matsize * matsize = %d * %d\n", matsize, matsize);
	printMat( "matC", matC, matsize, matsize );
	printMat( "matA", matA, matsize, matsize );
	printMat( "matB", matB, matsize, matsize );
	delete[] matA;
	delete[] matB;
	delete[] matC;

	return 0;
}
