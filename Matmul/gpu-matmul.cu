#include "hip/hip_runtime.h"
#include "common.c"

int matsize = 100;

__global__ void kernelMatMul( float* C, float* A, float* B, int matsize ) {
	int gy = blockIdx.y * blockDim.y + threadIdx.y; 
	int gx = blockIdx.x * blockDim.x + threadIdx.x; 
	if (gy < matsize && gx < matsize)
	{
		float sum = 0.0f;
		for (int k = 0; k < matsize; ++k) {
			int idxA = gy * matsize + k;
			int idxB = k * matsize + gx;
			sum += A[idxA] * B[idxB];
		}
		int idxC = gy * matsize + gx;
		C[idxC] = sum;
	}
}
int main(void) {
    float matA[matsize * matsize];
    float matB[matsize * matsize];
    float matC[matsize * matsize];
	srand( 0 );
	setNormalizedRandomData( matA, matsize * matsize );
	setNormalizedRandomData( matB, matsize * matsize );
	float* dev_matA = NULL;
	float* dev_matB = NULL;
	float* dev_matC = NULL;
	hipMalloc( (void**)&dev_matA,  matsize *  matsize * sizeof(float) );
	hipMalloc( (void**)&dev_matB,  matsize *  matsize * sizeof(float) );
	hipMalloc( (void**)&dev_matC,  matsize *  matsize * sizeof(float) );
	hipMemcpy( dev_matA, matA,  matsize * matsize * sizeof(float),  hipMemcpyHostToDevice);
	hipMemcpy( dev_matB, matB,  matsize * matsize * sizeof(float), hipMemcpyHostToDevice);
	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid((matsize + dimBlock.x - 1) / dimBlock.x, (matsize + dimBlock.y - 1) / dimBlock.y, 1);
	clock_t start = clock();
	kernelMatMul <<< dimGrid, dimBlock>>>( dev_matC, dev_matA, dev_matB, matsize);
	hipDeviceSynchronize();
	clock_t end = clock();
	hipMemcpy( matC, dev_matC, matsize *matsize * sizeof(float), hipMemcpyDeviceToHost);
	hipFree( dev_matA );
	hipFree( dev_matB );
	hipFree( dev_matC );
    double execution_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Execution time: %d usec\n", (int) (execution_time * 1000000));
	printf("matrix size = matsize * matsize = %d * %d\n", matsize, matsize);
	printMat( "matC", matC, matsize, matsize );
	printMat( "matA", matA, matsize, matsize );
	printMat( "matB", matB, matsize, matsize );
	return 0;
}
