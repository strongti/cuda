#include "hip/hip_runtime.h"
#include "./common.cpp"

int nrow = 100; 
int ncol = 100; 

__global__ void kernel_matadd( float* matC, float* matA, float* matB, int nrow, int ncol ) {
	int col = blockIdx.x * blockDim.x + threadIdx.x; 
	int row = blockIdx.y * blockDim.y + threadIdx.y; 
	//No for loop

	{
		//Matrix Add concept


	}
}

int main(void) {
	float* matA = nullptr;
	float* matB = nullptr;
	float* matC = nullptr;
	matA = new float[nrow * ncol];
	matB = new float[nrow * ncol];
	matC = new float[nrow * ncol];
	srand( 0 );
	setNormalizedRandomData( matA, nrow * ncol );
	setNormalizedRandomData( matB, nrow * ncol );
	float* dev_matA = nullptr;
	float* dev_matB = nullptr;
	float* dev_matC = nullptr;
	//Memory allocation to GPU



	//Memory Copy CPU to GPU


	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid((ncol + dimBlock.x - 1) / dimBlock.x, (nrow + dimBlock.y - 1) / dimBlock.y, 1);
	ELAPSED_TIME_BEGIN(0);
	kernel_matadd <<< dimGrid, dimBlock>>>( dev_matC, dev_matA, dev_matB, nrow, ncol );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	//Memory Copy GPU to CPU

	//Delete GPU's Memory


	CUDA_PRINT_CONFIG_2D(ncol ,nrow);
	printf("matrix size = nrow * ncol = %d * %d\n", nrow, ncol);
	printMat( "matC", matC, nrow, ncol );
	printMat( "matA", matA, nrow, ncol );
	printMat( "matB", matB, nrow, ncol );
	delete[] matA;
	delete[] matB;
	delete[] matC;
	return 0;
}
