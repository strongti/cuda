#include "hip/hip_runtime.h"
#include "common.c"


float alpha = 0.5f;
float beta = -100.0f;
int matsize = 100; 


__global__ void kernelGEMM( float* C, float* A, float* B, int matsize, float alpha, float beta ) {
	//Matrix multiply concept with Matrix add



	{








	}
}

int main(void) {
    float matA[matsize * matsize];
    float matB[matsize * matsize];
    float matC[matsize * matsize];
	printMat( "Before_matC", matC, matsize, matsize );
	srand( 0 );
	setNormalizedRandomData( matA, matsize * matsize );
	setNormalizedRandomData( matB, matsize * matsize );
	float* dev_matA = NULL;
	float* dev_matB = NULL;
	float* dev_matC = NULL;
	//Memory allocation to GPU



	//Memory Copy CPU to GPU



	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid((matsize + dimBlock.x - 1) / dimBlock.x, (matsize + dimBlock.y - 1) / dimBlock.y, 1);
	clock_t start = clock();
	kernelGEMM <<< dimGrid, dimBlock>>>(dev_matC, dev_matA, dev_matB, matsize, alpha, beta );
	hipDeviceSynchronize();
	clock_t end = clock();
	//Memory Copy GPU to CPU

	//Delete GPU's Memory



	double execution_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Execution time: %d usec\n", (int) (execution_time * 1000000));
	printf("matrix size = matsize * matsize = %d * %d\n", matsize, matsize);
	printMat( "matA", matA, matsize, matsize );
	printMat( "matB", matB, matsize, matsize );
	printMat( "After_matC", matC, matsize, matsize );
	return 0;
}
