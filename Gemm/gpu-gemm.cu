#include "hip/hip_runtime.h"
#include "./common.cpp"


float alpha = 0.5f;
float beta = -100.0f;
int matsize = 100; 


__global__ void kernelGEMM( float* C, float* A, float* B, int matsize, float alpha, float beta ) {
	//Matrix multiply concept with Matrix add



	{








	}
}

int main(void) {
	float* matA = nullptr;
	float* matB = nullptr;
	float* matC = nullptr;
	matA = new float[matsize * matsize];
	matB = new float[matsize * matsize];
	matC = new float[matsize * matsize];
	printMat( "Before_matC", matC, matsize, matsize );
	srand( 0 );
	setNormalizedRandomData( matA, matsize * matsize );
	setNormalizedRandomData( matB, matsize * matsize );
	float* dev_matA = nullptr;
	float* dev_matB = nullptr;
	float* dev_matC = nullptr;
	//Memory allocation to GPU



	//Memory Copy CPU to GPU



	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid((matsize + dimBlock.x - 1) / dimBlock.x, (matsize + dimBlock.y - 1) / dimBlock.y, 1);
	ELAPSED_TIME_BEGIN(0);
	kernelGEMM <<< dimGrid, dimBlock>>>(dev_matC, dev_matA, dev_matB, matsize, alpha, beta );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	//Memory Copy GPU to CPU

	//Delete GPU's Memory



	printf("matrix size = matsize * matsize = %d * %d\n", matsize, matsize);
	printMat( "matA", matA, matsize, matsize );
	printMat( "matB", matB, matsize, matsize );
	printMat( "After_matC", matC, matsize, matsize );
	delete[] matA;
	delete[] matB;
	delete[] matC;
	return 0;
}
