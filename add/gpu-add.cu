#include "hip/hip_runtime.h"
#include "common.c"

__global__ void add_kernel( int* c, int* a, int* b ) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main(void) {
	int SIZE = 5;
	int a[SIZE] = { 1, 2, 3, 4, 5 };
	int b[SIZE] = { 10, 20, 30, 40, 50 };
	int c[SIZE] = { 0 };
	int* dev_a = NULL;
	int* dev_b = NULL;
	int* dev_c = NULL;
	hipMalloc( (void**)&dev_a, SIZE * sizeof(int) );
	hipMalloc( (void**)&dev_b, SIZE * sizeof(int) );
	hipMalloc( (void**)&dev_c, SIZE * sizeof(int) );
	hipMemcpy( dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice );
	clock_t start = clock();
	add_kernel<<<1,SIZE>>>( dev_c, dev_a, dev_b );
	hipDeviceSynchronize();
	clock_t end = clock();
	hipMemcpy( c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost );
	hipFree(dev_a );
	hipFree(dev_b );
	hipFree(dev_c );
 	double execution_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Execution time: %d usec\n", (int) (execution_time * 1000000));
	printf("{%d,%d,%d,%d,%d} + {%d,%d,%d,%d,%d} = {%d,%d,%d,%d,%d}\n",
	       a[0], a[1], a[2], a[3], a[4],
	       b[0], b[1], b[2], b[3], b[4],
	       c[0], c[1], c[2], c[3], c[4]);
	return 0;
}