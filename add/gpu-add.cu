#include "hip/hip_runtime.h"
#include "./common.cpp"

__global__ void add_kernel( int* c, int* a, int* b ) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main(void) {
	int SIZE = 5;
	int a[SIZE] = { 1, 2, 3, 4, 5 };
	int b[SIZE] = { 10, 20, 30, 40, 50 };
	int c[SIZE] = { 0 };
	int* dev_a = nullptr;
	int* dev_b = nullptr;
	int* dev_c = nullptr;
	hipMalloc( (void**)&dev_a, SIZE * sizeof(int) );
	hipMalloc( (void**)&dev_b, SIZE * sizeof(int) );
	hipMalloc( (void**)&dev_c, SIZE * sizeof(int) );
	hipMemcpy( dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice );
	ELAPSED_TIME_BEGIN(0);
	add_kernel<<<1,SIZE>>>( dev_c, dev_a, dev_b );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	hipMemcpy( c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost );
	hipFree(dev_a );
	hipFree(dev_b );
	hipFree(dev_c );
	printf("{%d,%d,%d,%d,%d} + {%d,%d,%d,%d,%d} = {%d,%d,%d,%d,%d}\n",
	       a[0], a[1], a[2], a[3], a[4],
	       b[0], b[1], b[2], b[3], b[4],
	       c[0], c[1], c[2], c[3], c[4]);
	return 0;
}

